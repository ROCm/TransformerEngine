/*************************************************************************
 * This file was modified for portability to AMDGPU
 * Copyright (c) 2023-2024, Advanced Micro Devices, Inc. All rights reserved.
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

#ifndef USE_ROCM
size_t get_cublasLt_version() { return cublasLtGetVersion(); }

size_t get_cudnn_version() { return hipdnnGetVersion(); }
#endif
void placeholder() {}  // TODO(ksivamani) clean this up
